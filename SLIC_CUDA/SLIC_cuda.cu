#include "hip/hip_runtime.h"
//SLIC cuda kernel 


#include "SLIC_cuda.h"
#define MAX_DIST FLT_MAX
#define NNEIGH 3


#if __CUDA_ARCH__<300
texture<uchar4, hipTextureType2D, hipReadModeElementType> frameBGRA_tex;
surface<void, cudaSurfaceType2D> frameLab_surf;
surface<void, cudaSurfaceType2D> labels_surf;
#endif


//======== device local function ============

__device__ float2 operator-(const float2 & a, const float2 & b) { return make_float2(a.x - b.x, a.y - b.y); }
__device__ float3 operator-(const float3 & a, const float3 & b) { return make_float3(a.x - b.x, a.y - b.y, a.z - b.z); }
__device__ int2 operator+(const int2 & a, const int2 & b) { return make_int2(a.x + b.x, a.y + b.y); }

__device__ float computeDistance(float2 c_p_xy, float3 c_p_Lab, float areaSpx, float wc2){

	float ds2 = pow(c_p_xy.x, 2) + pow(c_p_xy.y, 2);
	float dc2 = pow(c_p_Lab.x, 2) + pow(c_p_Lab.y, 2) + pow(c_p_Lab.z, 2);
	float dist = sqrt(dc2 + ds2 / areaSpx*wc2);

	return dist;
}

__device__ int convertIdx(int2 wg, int lc_idx, int nBloc_per_row){

	int2 relPos2D = make_int2(lc_idx % 5 - 2, lc_idx / 5 - 2);
	int2 glPos2D = wg + relPos2D;


	return glPos2D.y*nBloc_per_row + glPos2D.x;
}

//============ Kernel ===============
#if __CUDA_ARCH__<300
__global__ void kRgb2CIELab(int width, int height)
{
	int px = blockIdx.x*blockDim.x + threadIdx.x;
	int py = blockIdx.y*blockDim.y + threadIdx.y;

	if (px < width && py < height) {
		uchar4 nPixel = tex2D(frameBGRA_tex, px, py);//inputImg[offset];

		float _b = (float)nPixel.x / 255.0;
		float _g = (float)nPixel.y / 255.0;
		float _r = (float)nPixel.z / 255.0;

		float x = _r * 0.412453 + _g * 0.357580 + _b * 0.180423;
		float y = _r * 0.212671 + _g * 0.715160 + _b * 0.072169;
		float z = _r * 0.019334 + _g * 0.119193 + _b * 0.950227;

		x /= 0.950456;
		float y3 = exp(log(y) / 3.0);
		z /= 1.088754;

		float l, a, b;

		x = x > 0.008856 ? exp(log(x) / 3.0) : (7.787 * x + 0.13793);
		y = y > 0.008856 ? y3 : 7.787 * y + 0.13793;
		z = z > 0.008856 ? z /= exp(log(z) / 3.0) : (7.787 * z + 0.13793);

		l = y > 0.008856 ? (116.0 * y3 - 16.0) : 903.3 * y;
		a = (x - y) * 500.0;
		b = (y - z) * 200.0;

		float4 fPixel;
		fPixel.x = l;
		fPixel.y = a;
		fPixel.z = b;
		fPixel.w = 0;

		surf2Dwrite(fPixel, frameLab_surf, px * 16, py);
	}
}
#else
__global__ void kRgb2CIELab(hipTextureObject_t inputImg, hipSurfaceObject_t outputImg, int width, int height)
{
	int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
	int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

	int px = blockIdx.x*blockDim.x + threadIdx.x;
	int py = blockIdx.y*blockDim.y + threadIdx.y;

	if (px<width && py<height) {
		uchar4 nPixel = tex2D<uchar4>(inputImg, px, py);//inputImg[offset];

		float _b = (float)nPixel.x / 255.0;
		float _g = (float)nPixel.y / 255.0;
		float _r = (float)nPixel.z / 255.0;

		float x = _r * 0.412453 + _g * 0.357580 + _b * 0.180423;
		float y = _r * 0.212671 + _g * 0.715160 + _b * 0.072169;
		float z = _r * 0.019334 + _g * 0.119193 + _b * 0.950227;

		x /= 0.950456;
		float y3 = exp(log(y) / 3.0);
		z /= 1.088754;

		float l, a, b;

		x = x > 0.008856 ? exp(log(x) / 3.0) : (7.787 * x + 0.13793);
		y = y > 0.008856 ? y3 : 7.787 * y + 0.13793;
		z = z > 0.008856 ? z /= exp(log(z) / 3.0) : (7.787 * z + 0.13793);

		l = y > 0.008856 ? (116.0 * y3 - 16.0) : 903.3 * y;
		a = (x - y) * 500.0;
		b = (y - z) * 200.0;

		float4 fPixel;
		fPixel.x = l;
		fPixel.y = a;
		fPixel.z = b;
		fPixel.w = 0;

		surf2Dwrite(fPixel, outputImg, px * 16, py);
	}

}
#endif

#if __CUDA_ARCH__<300
__global__ void k_initClusters(float* clusters, int width, int height, int nSpxPerRow, int nSpxPerCol){

	int idx_c = blockIdx.x*blockDim.x + threadIdx.x, idx_c5 = idx_c * 5;
	int nSpx = nSpxPerCol*nSpxPerRow;

	if (idx_c<nSpx){

		int wSpx = width / nSpxPerRow, hSpx = height / nSpxPerCol;

		int i = idx_c / nSpxPerRow;
		int j = idx_c%nSpxPerRow;

		int x = j*wSpx + wSpx / 2;
		int y = i*hSpx + hSpx / 2;

		float4 color;
		surf2Dread(&color, frameLab_surf, x * 16, y);


		clusters[idx_c5] = color.x;
		clusters[idx_c5 + 1] = color.y;
		clusters[idx_c5 + 2] = color.z;
		clusters[idx_c5 + 3] = x;
		clusters[idx_c5 + 4] = y;
	}
}

#else
__global__ void k_initClusters(hipSurfaceObject_t frameLab, float* clusters, int width, int height, int nSpxPerRow, int nSpxPerCol){

	int idx_c = blockIdx.x*blockDim.x + threadIdx.x, idx_c5 = idx_c * 5;
	int nSpx = nSpxPerCol*nSpxPerRow;

	if (idx_c<nSpx){

		int wSpx = width / nSpxPerRow, hSpx = height / nSpxPerCol;

		int i = idx_c / nSpxPerRow;
		int j = idx_c%nSpxPerRow;

		int x = j*wSpx + wSpx / 2;
		int y = i*hSpx + hSpx / 2;

		float4 color;
		surf2Dread(&color, frameLab, x * 16, y);


		clusters[idx_c5] = color.x;
		clusters[idx_c5 + 1] = color.y;
		clusters[idx_c5 + 2] = color.z;
		clusters[idx_c5 + 3] = x;
		clusters[idx_c5 + 4] = y;
	}
}
#endif

#if __CUDA_ARCH__<300
__global__ void k_assignement(int width, int height, int wSpx, int hSpx, float* clusters, float* accAtt_g, float wc2){

	// gather NNEIGH surrounding clusters

	__shared__ float4 sharedLab[NNEIGH][NNEIGH];
	__shared__ float2 sharedXY[NNEIGH][NNEIGH];

	int nClustPerRow = width / wSpx;
	int nn2 = NNEIGH / 2;


	if (threadIdx.x<NNEIGH && threadIdx.y<NNEIGH)
	{
		int id_x = threadIdx.x - nn2;
		int id_y = threadIdx.y - nn2;

		int clustLinIdx = blockIdx.x + id_y*nClustPerRow + id_x;
		if (clustLinIdx >= 0 && clustLinIdx<gridDim.x)
		{
			int clustLinIdx5 = clustLinIdx * 5;
			sharedLab[threadIdx.y][threadIdx.x].x = clusters[clustLinIdx5];
			sharedLab[threadIdx.y][threadIdx.x].y = clusters[clustLinIdx5 + 1];
			sharedLab[threadIdx.y][threadIdx.x].z = clusters[clustLinIdx5 + 2];

			sharedXY[threadIdx.y][threadIdx.x].x = clusters[clustLinIdx5 + 3];
			sharedXY[threadIdx.y][threadIdx.x].y = clusters[clustLinIdx5 + 4];
		}
		else
		{
			sharedLab[threadIdx.y][threadIdx.x].x = -1;
		}

	}

	__syncthreads();
	// Find nearest neighbour

	float areaSpx = wSpx*hSpx;
	float distanceMin = MAX_DIST;
	float labelMin = -1;

	int px_in_grid = blockIdx.x*blockDim.x + threadIdx.x;
	int py_in_grid = blockIdx.y*blockDim.y + threadIdx.y;

	int px = px_in_grid%width;

	if (py_in_grid<hSpx && px<width)
	{
		int py = py_in_grid + px_in_grid / width*hSpx;
		//int pxpy = py*width+px;

		float4 color;
		surf2Dread(&color, frameLab_surf, px * 16, py);

		//float3 px_Lab = make_float3(frameLab[pxpy].x,frameLab[pxpy].y,frameLab[pxpy].z);
		float3 px_Lab = make_float3(color.x, color.y, color.z);

		float2 px_xy = make_float2(px, py);

		for (int i = 0; i<NNEIGH; i++)
		{
			for (int j = 0; j<NNEIGH; j++)
			{
				if (sharedLab[i][j].x != -1)
				{
					float2 cluster_xy = make_float2(sharedXY[i][j].x, sharedXY[i][j].y);
					float3 cluster_Lab = make_float3(sharedLab[i][j].x, sharedLab[i][j].y, sharedLab[i][j].z);

					float2 px_c_xy = px_xy - cluster_xy;
					float3 px_c_Lab = px_Lab - cluster_Lab;

					float distTmp = fminf(computeDistance(px_c_xy, px_c_Lab, areaSpx, wc2), distanceMin);

					if (distTmp != distanceMin){
						distanceMin = distTmp;
						labelMin = blockIdx.x + (i - nn2)*nClustPerRow + (j - nn2);
					}

				}
			}
		}
		surf2Dwrite(labelMin, labels_surf, px * 4, py);

		int labelMin6 = int(labelMin * 6);
		atomicAdd(&accAtt_g[labelMin6], px_Lab.x);
		atomicAdd(&accAtt_g[labelMin6 + 1], px_Lab.y);
		atomicAdd(&accAtt_g[labelMin6 + 2], px_Lab.z);
		atomicAdd(&accAtt_g[labelMin6 + 3], px);
		atomicAdd(&accAtt_g[labelMin6 + 4], py);
		atomicAdd(&accAtt_g[labelMin6 + 5], 1); //counter
	}
}


#else
__global__ void k_assignement(int width, int height, int wSpx, int hSpx, hipSurfaceObject_t frameLab, hipSurfaceObject_t labels, float* clusters, float* accAtt_g, float wc2){

	// gather NNEIGH surrounding clusters

	__shared__ float4 sharedLab[NNEIGH][NNEIGH];
	__shared__ float2 sharedXY[NNEIGH][NNEIGH];

	int nClustPerRow = width / wSpx;
	int nn2 = NNEIGH / 2;


	if (threadIdx.x<NNEIGH && threadIdx.y<NNEIGH)
	{
		int id_x = threadIdx.x - nn2;
		int id_y = threadIdx.y - nn2;

		int clustLinIdx = blockIdx.x + id_y*nClustPerRow + id_x;
		if (clustLinIdx >= 0 && clustLinIdx<gridDim.x)
		{
			int clustLinIdx5 = clustLinIdx * 5;
			sharedLab[threadIdx.y][threadIdx.x].x = clusters[clustLinIdx5];
			sharedLab[threadIdx.y][threadIdx.x].y = clusters[clustLinIdx5 + 1];
			sharedLab[threadIdx.y][threadIdx.x].z = clusters[clustLinIdx5 + 2];

			sharedXY[threadIdx.y][threadIdx.x].x = clusters[clustLinIdx5 + 3];
			sharedXY[threadIdx.y][threadIdx.x].y = clusters[clustLinIdx5 + 4];
		}
		else
		{
			sharedLab[threadIdx.y][threadIdx.x].x = -1;
		}

	}

	__syncthreads();
	// Find nearest neighbour

	float areaSpx = wSpx*hSpx;
	float distanceMin = MAX_DIST;
	float labelMin = -1;

	int px_in_grid = blockIdx.x*blockDim.x + threadIdx.x;
	int py_in_grid = blockIdx.y*blockDim.y + threadIdx.y;

	int px = px_in_grid%width;

	if (py_in_grid<hSpx && px<width)
	{
		int py = py_in_grid + px_in_grid / width*hSpx;
		int pxpy = py*width + px;

		float4 color;
		surf2Dread(&color, frameLab, px * 16, py);

		//float3 px_Lab = make_float3(frameLab[pxpy].x,frameLab[pxpy].y,frameLab[pxpy].z);
		float3 px_Lab = make_float3(color.x, color.y, color.z);

		float2 px_xy = make_float2(px, py);

		for (int i = 0; i<NNEIGH; i++)
		{
			for (int j = 0; j<NNEIGH; j++)
			{
				if (sharedLab[i][j].x != -1)
				{
					float2 cluster_xy = make_float2(sharedXY[i][j].x, sharedXY[i][j].y);
					float3 cluster_Lab = make_float3(sharedLab[i][j].x, sharedLab[i][j].y, sharedLab[i][j].z);

					float2 px_c_xy = px_xy - cluster_xy;
					float3 px_c_Lab = px_Lab - cluster_Lab;

					float distTmp = fminf(computeDistance(px_c_xy, px_c_Lab, areaSpx, wc2), distanceMin);

					if (distTmp != distanceMin){
						distanceMin = distTmp;
						labelMin = blockIdx.x + (i - nn2)*nClustPerRow + (j - nn2);
					}

				}
			}
		}
		surf2Dwrite(labelMin, labels, px * 4, py);

		int labelMin6 = int(labelMin * 6);
		atomicAdd(&accAtt_g[labelMin6], px_Lab.x);
		atomicAdd(&accAtt_g[labelMin6 + 1], px_Lab.y);
		atomicAdd(&accAtt_g[labelMin6 + 2], px_Lab.z);
		atomicAdd(&accAtt_g[labelMin6 + 3], px);
		atomicAdd(&accAtt_g[labelMin6 + 4], py);
		atomicAdd(&accAtt_g[labelMin6 + 5], 1); //counter
	}
}
#endif


__global__ void k_update(int nSpx, float* clusters, float* accAtt_g)
{
	int cluster_idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (cluster_idx<nSpx)
	{
		uint cluster_idx6 = cluster_idx * 6;
		uint cluster_idx5 = cluster_idx * 5;
		int counter = accAtt_g[cluster_idx6 + 5];
		if (counter != 0){
			clusters[cluster_idx5] = accAtt_g[cluster_idx6] / counter;
			clusters[cluster_idx5 + 1] = accAtt_g[cluster_idx6 + 1] / counter;
			clusters[cluster_idx5 + 2] = accAtt_g[cluster_idx6 + 2] / counter;
			clusters[cluster_idx5 + 3] = accAtt_g[cluster_idx6 + 3] / counter;
			clusters[cluster_idx5 + 4] = accAtt_g[cluster_idx6 + 4] / counter;

			//reset accumulator
			accAtt_g[cluster_idx6] = 0;
			accAtt_g[cluster_idx6 + 1] = 0;
			accAtt_g[cluster_idx6 + 2] = 0;
			accAtt_g[cluster_idx6 + 3] = 0;
			accAtt_g[cluster_idx6 + 4] = 0;
			accAtt_g[cluster_idx6 + 5] = 0;
		}
	}
}

//============== wrapper =================

__host__ void SLIC_cuda::InitBuffers() {
	//allocate buffers on gpu
	//gpuErrchk(hipMalloc((void**)&frameBGRA_g, m_nPx*sizeof(uchar4))); //4 channels for padding

	hipChannelFormatDesc channelDescr = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	gpuErrchk(hipMallocArray(&frameBGRA_array, &channelDescr, m_width, m_height));

	hipChannelFormatDesc channelDescrLab = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	gpuErrchk(hipMallocArray(&frameLab_array, &channelDescrLab, m_width, m_height, hipArraySurfaceLoadStore));

	hipChannelFormatDesc channelDescrLabels = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	gpuErrchk(hipMallocArray(&labels_array, &channelDescrLabels, m_width, m_height, hipArraySurfaceLoadStore));
	//texture FrameBGR (read-only)



#if __CUDA_ARCH__>=300
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = frameBGRA_array;

	// Specify texture object parameters
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = false;
	gpuErrchk(hipCreateTextureObject(&frameBGRA_tex, &resDesc, &texDesc, NULL));
#else
	frameBGRA_tex.addressMode[0] = hipAddressModeClamp;
	frameBGRA_tex.addressMode[1] = hipAddressModeClamp;
	frameBGRA_tex.filterMode = hipFilterModePoint;
	frameBGRA_tex.normalized = false;
	hipBindTextureToArray(&frameBGRA_tex, frameBGRA_array, &channelDescr);
#endif

	// surface frameLab
#if __CUDA_ARCH__>=300
	hipResourceDesc resDescLab;
	memset(&resDescLab, 0, sizeof(resDescLab));
	resDescLab.resType = hipResourceTypeArray;

	resDescLab.res.array.array = frameLab_array;
	gpuErrchk(hipCreateSurfaceObject(&frameLab_surf, &resDescLab));
#else
	cudaBindSurfaceToArray(&frameLab_surf, frameLab_array, &channelDescrLab);
#endif

	// surface labels
#if __CUDA_ARCH__>=300

	hipResourceDesc resDescLabels;
	memset(&resDescLabels, 0, sizeof(resDescLabels));
	resDescLabels.resType = hipResourceTypeArray;

	resDescLabels.res.array.array = labels_array;
	gpuErrchk(hipCreateSurfaceObject(&labels_surf, &resDescLabels));

#else
	cudaBindSurfaceToArray(&labels_surf, labels_array, &channelDescrLabels);
#endif

	// buffers clusters , accAtt
	gpuErrchk(hipMalloc((void**)&clusters_g, m_nSpx*sizeof(float) * 5)); // 5-D centroid
	gpuErrchk(hipMalloc((void**)&accAtt_g, m_nSpx*sizeof(float) * 6)); // 5-D centroid acc + 1 counter
	hipMemset(accAtt_g, 0, m_nSpx*sizeof(float) * 6);//initialize accAtt to 0
}


#if __CUDA_ARCH__>=300
__host__ void SLIC_cuda::Rgb2CIELab(hipTextureObject_t inputImg, hipSurfaceObject_t outputImg, int width, int height)
{
	int side = 16;
	dim3 threadsPerBlock(side, side);
	dim3 numBlocks(iDivUp(m_width, side), iDivUp(m_height, side));
	kRgb2CIELab << <numBlocks, threadsPerBlock >> >(inputImg, outputImg, width, height);
}
#else
__host__ void SLIC_cuda::Rgb2CIELab(int width, int height)
{
	int side = 16;
	dim3 threadsPerBlock(side, side);
	dim3 numBlocks(iDivUp(m_width, side), iDivUp(m_height, side));
	kRgb2CIELab << <numBlocks, threadsPerBlock >> >(width, height);
}
#endif

__host__ void SLIC_cuda::InitClusters()
{
	dim3 threadsPerBlock(NMAX_THREAD);
	dim3 numBlocks(iDivUp(m_nSpx, NMAX_THREAD));
#if __CUDA_ARCH__>=300
	k_initClusters << <numBlocks, threadsPerBlock >> >(frameLab_surf, clusters_g, m_width, m_height, m_width / m_wSpx, m_height / m_hSpx);
#else
	k_initClusters << <numBlocks, threadsPerBlock >> >(clusters_g, m_width, m_height, m_width / m_wSpx, m_height / m_hSpx);
#endif
}
__host__ void SLIC_cuda::Assignement() {

	int hMax = NMAX_THREAD / m_hSpx;
	int nBlockPerClust = iDivUp(m_hSpx, hMax);

	dim3 blockPerGrid(m_nSpx, nBlockPerClust);
	dim3 threadPerBlock(m_wSpx, std::min(m_hSpx, hMax));

	CV_Assert(threadPerBlock.x >= 3 && threadPerBlock.y >= 3);

	float wc2 = m_wc * m_wc;
#if __CUDA_ARCH__>=300
	k_assignement << < blockPerGrid, threadPerBlock >> >(m_width, m_height, m_wSpx, m_hSpx, frameLab_surf, labels_surf, clusters_g, accAtt_g, wc2);
#else
	k_assignement << < blockPerGrid, threadPerBlock >> >(m_width, m_height, m_wSpx, m_hSpx, clusters_g, accAtt_g, wc2);

#endif
}
__host__ void SLIC_cuda::Update()
{
	dim3 threadsPerBlock(NMAX_THREAD);
	dim3 numBlocks(iDivUp(m_nSpx, NMAX_THREAD));
	k_update << <numBlocks, threadsPerBlock >> >(m_nSpx, clusters_g, accAtt_g);
}



